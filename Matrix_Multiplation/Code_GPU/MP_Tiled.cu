#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <chrono>

#define BLOCK_DIM 16

#define TILE_WIDTH 16

using namespace std::chrono;

void time_stats(float micro_seconds) {
    printf("Execution times:\n");
    printf("    * %.0f μs \n", micro_seconds);
    printf("    * %.2f ms \n", micro_seconds / 1000);
    printf("    * %.2f s \n", micro_seconds / 1000 / 1000);
    printf("\n");
}


__global__ void MatrixMulKernelTiled(float *mat_a, float *mat_b, float *res_mat, int size) {
    __shared__ float M_ds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float N_ds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    if ((Row < size) && (Col < size)) {
        float Pvalue = 0;
        for (int m = 0; m <= size / TILE_WIDTH; ++m) {
            M_ds[ty][tx] = mat_a[Row * size + m * TILE_WIDTH + tx];
            N_ds[ty][tx] = mat_b[(m * TILE_WIDTH + ty) * size + Col];

            __syncthreads();

            for (int k = 0; k < TILE_WIDTH; ++k) {
                Pvalue += M_ds[ty][k] * N_ds[k][tx];
            }
            __syncthreads();
        }

        res_mat[Row * size + Col] = Pvalue;
    }
}

int main(void) {
    int sizes[5] = {1024, 2048, 4096, 8192, 16384};

    float *mat_a_host, *mat_b_host, *mat_res_host_gpu;
    float *mat_a_dev, *mat_b_dev, *mat_res_dev;
    dim3 gridDim, blockDim;

    for (int i = 0; i < 5; i++) {
        long nBytes = sizes[i] * sizes[i] * sizeof(float);

        mat_a_host = (float*)malloc(nBytes);
        mat_b_host = (float*)malloc(nBytes);
        mat_res_host_gpu = (float*)malloc(nBytes);

        hipMalloc((void**)&mat_a_dev, nBytes);
        hipMalloc((void**)&mat_b_dev, nBytes);
        hipMalloc((void**)&mat_res_dev, nBytes);

        for (int j = 0; j < sizes[i] * sizes[i]; j++) {
            mat_a_host[j] = 1;
            mat_b_host[j] = 1;
        }

        hipMemcpy(mat_a_dev, mat_a_host, nBytes, hipMemcpyDefault);
        hipMemcpy(mat_b_dev, mat_b_host, nBytes, hipMemcpyDefault);
        hipMemset(mat_res_dev, 0, nBytes);

        blockDim.x = BLOCK_DIM;
        blockDim.y = BLOCK_DIM;
        gridDim.x = sizes[i] / blockDim.x + ((sizes[i] % blockDim.x) == 0 ? 0 : 1);
        gridDim.y = sizes[i] / blockDim.y + ((sizes[i] % blockDim.y) == 0 ? 0 : 1);

        auto start = high_resolution_clock::now();
        MatrixMulKernelTiled<<<gridDim, blockDim>>>(mat_a_dev, mat_b_dev, mat_res_dev, sizes[i]);
        hipDeviceSynchronize();
        auto stop = high_resolution_clock::now();

        hipMemcpy(mat_res_host_gpu, mat_res_dev, nBytes, hipMemcpyDeviceToHost);

        long check = 0;
        for (int k = 0; k < sizes[i] * sizes[i]; k++) {
            check += (long) mat_res_host_gpu[i];
        }

        printf("Matrix size: %d x %d \n", sizes[i], sizes[i]);
        printf("Block size: %d x %d = %d\n", BLOCK_DIM, BLOCK_DIM, BLOCK_DIM * BLOCK_DIM);
        printf("Check: %ld\n", check);
        time_stats(duration_cast<microseconds>(stop - start).count());

        free(mat_a_host);
        free(mat_b_host);

        hipFree(mat_a_dev);
        hipFree(mat_b_dev);
        hipFree(mat_res_dev);
    }

    return 0;
}