#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define SIZE 4
#define BLOCK_DIM 64

// Somma Seriale
__host__ float CPU_SUM(float *a, int n) {
    int i;
    float c = 0;

    for (i = 0; i < n; i++) c += a[i];

    return c;
}
// Somma Parallela
__global__ void GPU_SUM(float *a, float *b, float *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) c[index] = a[index] * b[index];
}
// Stampa vettore
void printVet(float *vet, int size) {
    printf("\n");
    for (int i = 0; i < size; i++) {
        printf("|%05.2f", vet[i]);
    }
    printf("|\n\n");
}

int main(void) {
    float res;
    float *a_host, *b_host, *res_host_gpu;
    float *a_dev, *b_dev, *c_dev;
    dim3 gridDim, blockDim;

    // Allocazione memoria
    int nBytes = SIZE * sizeof(float);
    a_host = (float *)malloc(nBytes);
    b_host = (float *)malloc(nBytes);
    res_host_gpu = (float *)calloc(nBytes, sizeof(float));
    hipMalloc((void **)&a_dev, nBytes);
    hipMalloc((void **)&b_dev, nBytes);
    hipMalloc((void **)&c_dev, nBytes);

    // Inizializzo i dati
    srand((unsigned int)time(0));
    for (int i = 0; i < SIZE; i++) {
        a_host[i] = rand() % 5 - 2;
        b_host[i] = rand() % 5 - 2;
    }

    // Copia dei dati dall'host al device
    hipMemcpy(a_dev, a_host, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b_host, nBytes, hipMemcpyHostToDevice);
    // Azzeriamo il contenuto del vettore c
    hipMemset(c_dev, 0, nBytes);

    // Configurazione del kernel
    blockDim.x = BLOCK_DIM;
    gridDim.x = SIZE / blockDim.x + ((SIZE % blockDim.x) == 0 ? 0 : 1);

    // Creazione eventi
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    GPU_SUM<<<gridDim, blockDim>>>(a_dev, b_dev, c_dev, SIZE);

    // Copia dei dati dall'device al host
    hipMemcpy(res_host_gpu, c_dev, nBytes, hipMemcpyDeviceToHost);

    // Calcolo valore finale su CPU
    res = CPU_SUM(res_host_gpu, SIZE);

    // Fine computazione su GPU
    hipEventRecord(stop);

    // Attende la terminazione di tutti thread
    hipEventSynchronize(stop);

    // Tempo tra i due eventi in millisecondi su GPU
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Tempo GPU-CPU=%f\n", elapsed);

    printf("\n");
    if (SIZE < 20) {
        printf("Vettore A:");
        printVet(a_host, SIZE);
        printf("Vettore B:");
        printVet(b_host, SIZE);
        printf("Vettore result della GPU:");
        printVet(res_host_gpu, SIZE);
    }
    printf("Prodotto finale: %.2f\n", res);

    free(a_host);
    free(b_host);
    free(res_host_gpu);

    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);

    return 0;
}