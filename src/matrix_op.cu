#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <omp.h>
#include <iostream>

#include "../include/matrix_op.cuh"

using namespace nvcuda;
//Codice dell'indiano ganesh
__global__ void mm_tiled_kernel(float * A, float * B, float * C,
                                    int size)
{
    __shared__ float sA[BLOCK_DIM][BLOCK_DIM];   // Tile size to store elements in shared memory
    __shared__ float sB[BLOCK_DIM][BLOCK_DIM];

    int Row = blockDim.y*blockIdx.y + threadIdx.y; //To generate ids of threads.
    int Col = blockDim.x*blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int k = 0; k < (((size - 1)/ BLOCK_DIM) + 1); k++)
    {
        if ( (Row < size) && (threadIdx.x + (k*BLOCK_DIM)) < size)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sA[threadIdx.y][threadIdx.x] = A[(Row*size) + threadIdx.x + (k*BLOCK_DIM)];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }
        if ( Col < size && (threadIdx.y + k*BLOCK_DIM) < size)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*BLOCK_DIM)*size + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < BLOCK_DIM; ++j)//Multiplying Elements present in tile
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    if (Row < size && Col < size)//Saving Final result into Matrix C
    {
        C[Row*size + Col] = Cvalue;
    }
}
// Kernels
/* GPU codice nostro
__global__ void mm_tiled_kernel(float* mat_a, float* mat_b, float* res_mat, int size) {
    __shared__ float m_a_sh[BLOCK_DIM][BLOCK_DIM];
    __shared__ float m_b_sh[BLOCK_DIM][BLOCK_DIM];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int mat_a_begin = size * BLOCK_DIM * by;
    int mat_a_end = mat_a_begin + size - 1;
    int mat_b_begin = BLOCK_DIM * bx;

    int mat_b_step = BLOCK_DIM * size;

    float temp = 0;
    for (int a = mat_a_begin, b = mat_b_begin; a <= mat_a_end; a += BLOCK_DIM, b += mat_b_step) {
        m_a_sh[ty][tx] = mat_a[a + size * ty + tx];
        m_b_sh[ty][tx] = mat_b[b + size * ty + tx];

        __syncthreads();

#pragma unroll
        for (int k = 0; k < BLOCK_DIM; ++k) {
            temp += m_a_sh[ty][k] * m_b_sh[k][tx];
        }

        __syncthreads();
    }

    int c = size * BLOCK_DIM * by + BLOCK_DIM * bx;
    res_mat[c + size * ty + tx] = temp;
}*/
__global__ void mm_tensor_kernel(half* mat_a, half* mat_b, float* res_mat, int size) {
    // Tile using a 2D grid
    int tile_row = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int tile_col = (blockIdx.y * blockDim.y + threadIdx.y);

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // Parte da 0 e sale di 16 alla volta
    for (int i = 0; i < size; i += WMMA_K) {
        int aCol = i;  // 0
        int aRow = tile_row * WMMA_M;

        int bCol = tile_col * WMMA_N;
        int bRow = i;  // 0

        // Bounds checking
        if (aRow < size && aCol < size && bRow < size && bCol < size) {
            // Load the inputs
            wmma::load_matrix_sync(a_frag, mat_a + (aRow * size) + aCol, size);  // mat_a[aRow, aCol]
            wmma::load_matrix_sync(b_frag, mat_b + (bRow * size) + bCol, size);  // mat_b[bRow, bCol]

            // Perform the matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    int cCol = tile_row * WMMA_N;
    int cRow = tile_col * WMMA_M;

    // Store the output
    wmma::store_matrix_sync(res_mat + (cRow * size) + cCol, acc_frag, size, wmma::mem_row_major);  // mat_c[cRow, cCol]
}

// Functions
double mm_cpu(float* mat_a, float* mat_b, float* mat_res, int size) {
    double t_init = omp_get_wtime();

//Loop interchange tra k_row e mat_col
//#pragma omp parallel for collapse(3)
    for (int mat_row = 0; mat_row < size; mat_row++)
            for (int k_row = 0; k_row < size; k_row++)
        for (int mat_col = 0; mat_col < size; mat_col++)
                mat_res[mat_row * size + mat_col] += mat_a[mat_row * size + k_row] * mat_b[k_row * size + mat_col];

    return omp_get_wtime() - t_init;
}
double mm_gpu(float* mat_a, float* mat_b, float* mat_res, int size) {
    float *res_mat_dev, *mat_a_dev, *mat_b_dev;

    hipMalloc((void**)&mat_a_dev, size * size * sizeof(float));
    hipMalloc((void**)&mat_b_dev, size * size * sizeof(float));
    hipMalloc((void**)&res_mat_dev, size * size * sizeof(float));

    hipMemcpy(mat_a_dev, mat_a, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(mat_b_dev, mat_b, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(res_mat_dev, 0, size * size * sizeof(float));

    dim3 gridDim, blockDim;

    blockDim.x = BLOCK_DIM;
    blockDim.y = BLOCK_DIM;
    gridDim.x = size / blockDim.x + ((size % blockDim.x) == 0 ? 0 : 1);
    gridDim.y = size / blockDim.y + ((size % blockDim.y) == 0 ? 0 : 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mm_tiled_kernel<<<gridDim, blockDim>>>(mat_a_dev, mat_b_dev, res_mat_dev, size);
    hipEventRecord(stop);

    hipMemcpy(mat_res, res_mat_dev, size * size * sizeof(float), hipMemcpyDeviceToHost);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    hipFree(mat_a_dev);
    hipFree(mat_b_dev);
    hipFree(res_mat_dev);

    return elapsed;
}
double mm_tensor(half* mat_a, half* mat_b, float* mat_res, int size) {
    half *mat_b_dev, *mat_a_dev;
    float* res_mat_dev;

    hipMalloc((void**)&mat_a_dev, size * size * sizeof(half));
    hipMalloc((void**)&mat_b_dev, size * size * sizeof(half));
    hipMalloc((void**)&res_mat_dev, size * size * sizeof(float));

    hipMemcpy(mat_a_dev, mat_a, size * size * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(mat_b_dev, mat_b, size * size * sizeof(half), hipMemcpyHostToDevice);
    hipMemset(res_mat_dev, 0, size * size * sizeof(float));

    dim3 gridDim, blockDim;

    blockDim.x = 128;
    blockDim.y = 4;
    gridDim.x = (size + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
    gridDim.y = (size + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mm_tensor_kernel<<<gridDim, blockDim>>>(mat_a_dev, mat_b_dev, res_mat_dev, size);
    hipEventRecord(stop);

    hipMemcpy(mat_res, res_mat_dev, size * size * sizeof(float), hipMemcpyDeviceToHost);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    hipFree(mat_a_dev);
    hipFree(mat_b_dev);
    hipFree(res_mat_dev);

    return elapsed;
}

// Matrix Checker, abbiamo sostituito mat_res[i]!=16 con !=size
bool mm_checker(float* mat_res, int size) {
    for (int i = 0; i < size * size; i++)
        if (mat_res[i] != size)
            return false;
    return true;
}