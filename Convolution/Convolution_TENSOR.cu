#include <assert.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>

#define SIZE 32
#define MASK_SIZE 16

#define PRINT_GREEN(str) printf("\x1b[32m%s\x1b[0m", str);
#define PRINT_RED(str) printf("\x1b[31m%s\x1b[0m", str);

// MMA matrix tile dimensions.
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

#define debug_x 0
#define debug_y 0

using namespace nvcuda;

void time_stats(float micro_seconds) {
    printf("Execution times:\n");
    printf("    * %.0f μs \n", micro_seconds);
    printf("    * %.2f ms \n", micro_seconds / 1000);
    printf("    * %.2f s \n", micro_seconds / 1000 / 1000);
    printf("\n");
}

void printMat(half* mat, int m, int n) {
    for (int i = 0; i < m * n; i++) {
        printf("|");
        printf(" %04.0f ", __half2float(mat[i]));
        if (((i + 1) % (n) == 0) && (i != 0))
            printf("|\n");
        if ((m * n) == 1)
            printf("|\n");
        if (n == 1 && ((i == 0)))
            printf("|\n");
    }
    printf("\n");
}

__global__ void WMMAF16TensorCore(half* mat, half* mask, half* mat_res_temp, int size) {
    int tile_row = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int tile_col = (blockIdx.y * blockDim.y + threadIdx.y);

    if (threadIdx.x == debug_x && threadIdx.y == debug_y) {
        printf("tile_row: %d\n", tile_row);
        printf("tile_col: %d\n", tile_col);
        printf("\n");
    }

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> mat_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> mask_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> acc_frag;
    wmma::fill_fragment(acc_frag, 0.0f);

    // Charge mask fragment
    wmma::load_matrix_sync(mask_frag, mask, MASK_SIZE);  // mask[bRow, bCol]

    int aCol = tile_col * 16;
    int aRow = tile_row * 16;

    for (int col = tile_col * MASK_SIZE; col < MASK_SIZE + (tile_col * MASK_SIZE); col++) {
        for (int row = tile_row * MASK_SIZE; row < MASK_SIZE + (tile_row * MASK_SIZE); row++) {

            if (aRow < size && aCol < size) {
                if (threadIdx.x == debug_x && threadIdx.y == debug_y) {
                    printf("Coordinate A [%d,%d]\n", aRow, aCol);
                    printf("Coordinate A [%d,%d]\n", aRow, aCol);
                }

                // Load the inputs
                wmma::load_matrix_sync(mat_frag, mat + (aRow * size) + aCol, MASK_SIZE);

                // Perform the matrix multiplication
                wmma::mma_sync(acc_frag, mat_frag, mask_frag, acc_frag);

                // Store the output
                // wmma::store_matrix_sync(mat_res_temp, acc_frag, MASK_SIZE, wmma::mem_col_major);  // mat_res_temp[cRow, cCol]
                wmma::store_matrix_sync(mat_res_temp + (aRow * size) + aCol, acc_frag, size, wmma::mem_col_major);  // mat_res_temp[cRow, cCol]

                int tot = 0;
                for (int i = 0; i < MASK_SIZE; i++) {
                    tot += __half2int_rd(mat_res_temp[i * MASK_SIZE + i]);
                }

                if (threadIdx.x == debug_x && threadIdx.y == debug_y) {
                    printf("tot: %d\n", tot);
                    // if (threadIdx.x == debug_x && threadIdx.y == debug_y) {
                    //     for (int i = 0; i < MASK_SIZE * MASK_SIZE; i++) {
                    //         printf("|");
                    //         printf(" %06.1f ", __half2float(mat_res_temp[i]));
                    //         if (((i + 1) % (MASK_SIZE) == 0) && (i != 0))
                    //             printf("|\n");
                    //         if ((MASK_SIZE * MASK_SIZE) == 1)
                    //             printf("|\n");
                    //         if (MASK_SIZE == 1 && ((i == 0)))
                    //             printf("|\n");
                    //     }
                    //     printf("\n");
                    // }
                }
            }
        }
    }
}

int main(void) {
    half *mat_host, *mask_host;
    half* mat_res_host_gpu;
    half *mat_dev, *mask_dev;
    half* mat_res_dev;
    dim3 gridDim, blockDim;

    mat_host = (half*)malloc(SIZE * SIZE * sizeof(half));
    mask_host = (half*)malloc(MASK_SIZE * MASK_SIZE * sizeof(half));
    mat_res_host_gpu = (half*)malloc(SIZE * SIZE * sizeof(half));

    hipMalloc((void**)&mat_dev, SIZE * SIZE * sizeof(half));
    hipMalloc((void**)&mask_dev, MASK_SIZE * MASK_SIZE * sizeof(half));
    hipMalloc((void**)&mat_res_dev, SIZE * SIZE * sizeof(half));

    float k = 0;
    for (int j = 0; j < SIZE * SIZE; j++) {
        mat_host[j] = __float2half(k);
        k += 0.025;
    }

    k = 0;
    for (int j = 0; j < MASK_SIZE * MASK_SIZE; j++) {
        mask_host[j] = __float2half(k);
        k += 0.025;
    }

    // printf("Mat A: \n");
    // printMat(mat_host, SIZE, SIZE);
    // printf("Mat B: \n");
    // printMat(mask_host, MASK_SIZE, MASK_SIZE);

    hipMemcpy(mat_dev, mat_host, SIZE * SIZE * sizeof(half), hipMemcpyDefault);
    hipMemcpy(mask_dev, mask_host, MASK_SIZE * MASK_SIZE * sizeof(half), hipMemcpyDefault);
    hipMemset(mat_res_dev, 0, SIZE * SIZE * sizeof(half));

    blockDim.x = 128;
    blockDim.y = 4;
    gridDim.x = (SIZE + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
    gridDim.y = (SIZE + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

    // printf("Griglia: %d, %d\n", gridDim.x, gridDim.y);
    // printf("Blocco: %d, %d\n", blockDim.x, blockDim.y);
    // printf("\n");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    WMMAF16TensorCore<<<gridDim, blockDim>>>(mat_dev, mask_dev, mat_res_dev, SIZE);
    hipEventRecord(stop);

    hipMemcpy(mat_res_host_gpu, mat_res_dev, SIZE * SIZE * sizeof(half), hipMemcpyDeviceToHost);

    // printf("Matrix size: %d x %d \n", sizes[i], sizes[i]);
    // printf("Check: ");
    // if (check) {
    //     PRINT_GREEN("Verified\n");
    // } else {
    //     PRINT_RED("Error\n");
    // }

    // printf("\nRisultato:\n");
    // printMat(mat_res_host_gpu, SIZE, SIZE);

    free(mat_host);
    free(mask_host);

    hipFree(mat_dev);
    hipFree(mask_dev);
    hipFree(mat_res_dev);

    return 0;
}