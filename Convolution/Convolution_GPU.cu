#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <chrono>

#define MASK_SIZE 4
#define MASK_CENTER 2

#define BLOCK_DIM 32
#define TILE_WIDTH 32

using namespace std;
using namespace std::chrono;

void time_stats(float micro_seconds) {
    printf("Execution times:\n");
    printf("    * %.0f μs \n", micro_seconds * 1000);
    printf("    * %.2f ms \n", micro_seconds);
    printf("    * %.2f s \n", micro_seconds / 1000);
    printf("\n");
}

void printMat(float* mat, int size) {
    // Print the entire matrix
    printf("\n");
    for (int i = 0; i < (size * size); i++) {
        printf("|");
        printf("%05.2f", mat[i]);
        if (((i + 1) % (size) == 0) && (i != 0))
            printf("|\n");
        if ((size * size) == 1)
            printf("|\n");
        if (size == 1 && ((i == 0)))
            printf("|\n");
    }
    printf("\n");
}

__global__ void ConvolutionKernel(float* mat_start, float* mat_res, const float* Mask, int size) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Coordinate result
    int row_o = blockIdx.y * TILE_WIDTH + ty;
    int col_o = blockIdx.x * TILE_WIDTH + tx;

    // Coordinate start
    int row_i = row_o - MASK_CENTER;
    int col_i = col_o - MASK_CENTER;

    // Tile in shared memory
    __shared__ float n_ds[TILE_WIDTH + MASK_SIZE * MASK_SIZE - 1][TILE_WIDTH + MASK_SIZE * MASK_SIZE - 1];

    // Tile cooperative upload
    if ((row_i >= 0) && (row_i < size) && (col_i >= 0) && (col_i < size)) {
        n_ds[ty][tx] = mat_start[(row_i * size) + col_i];
    }

     __syncthreads();

    // Convolution calculation
    float output = 0.0f;
    if (ty < TILE_WIDTH && tx < TILE_WIDTH) {
        for (int i = 0; i < MASK_SIZE; i++) {
            for (int j = 0; j < MASK_SIZE; j++) {
                output += Mask[(i * MASK_SIZE) + j] * n_ds[i + ty][j + tx];
            }
        }
        if (row_o < size && col_o < size) {
            mat_res[row_o * size + col_o] = output;
        }
    }
}

int main() {
    int sizes[5] = {16, 16, 16, 16, 16};

    float *mat_start_host, *mat_res_host, *mask_host;
    float *mat_start_dev, *mat_res_dev, *mask_dev;

    // Mask init and upload
    mask_host = (float*)malloc(MASK_SIZE * MASK_SIZE * sizeof(float));
    for (int i = 0; i < MASK_SIZE * MASK_SIZE; i++) {
        mask_host[i] = 1;
    }
    hipMalloc((void**)&mask_dev, MASK_SIZE * MASK_SIZE * sizeof(float));
    hipMemcpy(mask_dev, mask_host, MASK_SIZE * MASK_SIZE * sizeof(float), hipMemcpyDefault);

    dim3 gridDim, blockDim;

    for (int k = 0; k < 5; k++) {
        mat_start_host = (float*)malloc(sizes[k] * sizes[k] * sizeof(float));
        mat_res_host = (float*)calloc(sizes[k] * sizes[k], sizeof(float));

        for (int i = 0; i < sizes[k] * sizes[k]; i++) {
            mat_start_host[i] = 1;
        }

        hipMalloc((void**)&mat_start_dev, sizes[k] * sizes[k] * sizeof(float));
        hipMalloc((void**)&mat_res_dev, sizes[k] * sizes[k] * sizeof(float));

        hipMemcpy(mat_start_dev, mat_start_host, sizes[k] * sizes[k] * sizeof(float), hipMemcpyDefault);
        hipMemcpy(mat_res_dev, mat_res_host, sizes[k] * sizes[k] * sizeof(float), hipMemcpyDefault);
        hipMemset(mat_res_dev, 0, sizes[k] * sizes[k] * sizeof(float));

        blockDim.x = BLOCK_DIM;
        blockDim.y = BLOCK_DIM;
        gridDim.x = sizes[k] / blockDim.x + ((sizes[k] % blockDim.x) == 0 ? 0 : 1);
        gridDim.y = sizes[k] / blockDim.y + ((sizes[k] % blockDim.y) == 0 ? 0 : 1);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        ConvolutionKernel<<<gridDim, blockDim>>>(mat_start_dev, mat_res_dev, mask_dev, sizes[k]);
        hipEventRecord(stop);

        hipMemcpy(mat_res_host, mat_res_dev, sizes[k] * sizes[k] * sizeof(float), hipMemcpyDeviceToHost);

        printMat(mat_res_host, sizes[k]);

        printf("Matrix size: %d x %d \n", sizes[k], sizes[k]);
        printf("Block size: %d x %d = %d\n", BLOCK_DIM, BLOCK_DIM, BLOCK_DIM * BLOCK_DIM);
        
        float elapsed;
        hipEventElapsedTime(&elapsed, start, stop);
        time_stats(elapsed);

        free(mat_start_host);
        free(mat_res_host);

        hipFree(mat_start_dev);
        hipFree(mat_res_dev);
    }
    free(mask_host);
    hipFree(mask_dev);
    
    return 0;
}