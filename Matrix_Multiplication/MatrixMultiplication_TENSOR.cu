#include <assert.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>

#define PRINT_GREEN(str) printf("\x1b[32m%s\x1b[0m", str);
#define PRINT_RED(str) printf("\x1b[31m%s\x1b[0m", str);

// MMA matrix tile dimensions.
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

using namespace nvcuda;

void time_stats(float micro_seconds) {
    printf("Execution times:\n");
    printf("    * %.0f μs \n", micro_seconds);
    printf("    * %.2f ms \n", micro_seconds / 1000);
    printf("    * %.2f s \n", micro_seconds / 1000 / 1000);
    printf("\n");
}

void printMat(float* mat, int size) {
    // Print the entire matrix
    printf("\n");
    for (int i = 0; i < (size * size); i++) {
        printf("|");
        printf("%05.2f", mat[i]);
        if (((i + 1) % (size) == 0) && (i != 0))
            printf("|\n");
        if ((size * size) == 1)
            printf("|\n");
        if (size == 1 && ((i == 0)))
            printf("|\n");
    }
    printf("\n");
}

__global__ void WMMAF16TensorCore(half* mat_a, half* mat_b, float* mat_c, int size) {
    // int row = blockIdx.y * blockDim.y + threadIdx.y;
    // int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Tile using a 2D grid
    int tile_row = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int tile_col = (blockIdx.y * blockDim.y + threadIdx.y);


    
    printf("Block_Dim: [%d,%d], Block_Thread: [%d,%d], Coord_Thread: [%d,%d], Tile M/N: [%d,%d]\n", blockDim.x, blockDim.y, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, tile_row, tile_col);
     

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // Parte da 0 e sale di 16 alla volta
    for (int i = 0; i < size; i += WMMA_K) {
        int aCol = i;  // 0
        int aRow = tile_row * WMMA_M;

        int bCol = tile_col * WMMA_N;
        int bRow = i;  // 0

        // Bounds checking
        if (aRow < size && aCol < size && bRow < size && bCol < size) {
            // printf("Block_Thread: [%d,%d], Coord_Thread: [%d,%d], A[%d,%d], B[%d,%d], ID: %ld, IDM: %ld, I:%d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, aRow, aCol, bRow, bCol, mat_a, i);
            // Load the inputs
            wmma::load_matrix_sync(a_frag, mat_a + (aRow * size) + aCol, size);  // mat_a[aRow, aCol]
            wmma::load_matrix_sync(b_frag, mat_b + (bRow * size) + bCol, size);  // mat_b[bRow, bCol]

            // Perform the matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    int cCol = tile_row * WMMA_N;
    int cRow = tile_col * WMMA_M;

    // Store the output
    wmma::store_matrix_sync(mat_c + (cRow * size) + cCol, acc_frag, size, wmma::mem_row_major);  // mat_c[cRow, cCol]
}

int main(void) {
    int sizes[1] = {32};

    half *mat_a_host, *mat_b_host;
    float* mat_res_host_gpu;
    half *mat_a_dev, *mat_b_dev;
    float* mat_res_dev;
    dim3 gridDim, blockDim;

    for (int i = 0; i < 1; i++) {
        long nBytes = sizes[i] * sizes[i] * sizeof(float);

        mat_a_host = (half*)malloc(nBytes);
        mat_b_host = (half*)malloc(nBytes);
        mat_res_host_gpu = (float*)malloc(nBytes);

        hipMalloc((void**)&mat_a_dev, nBytes);
        hipMalloc((void**)&mat_b_dev, nBytes);
        hipMalloc((void**)&mat_res_dev, nBytes);

        for (int j = 0; j < sizes[i] * sizes[i]; j++) {
            mat_a_host[j] = __float2half(1);
            mat_b_host[j] = __float2half(1);
        }

        hipMemcpy(mat_a_dev, mat_a_host, nBytes, hipMemcpyDefault);
        hipMemcpy(mat_b_dev, mat_b_host, nBytes, hipMemcpyDefault);
        hipMemset(mat_res_dev, 0, nBytes);

        blockDim.x = 128;
        blockDim.y = 4;
        gridDim.x = (sizes[i] + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
        gridDim.y = (sizes[i] + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

        printf("Griglia: %d, %d\n", gridDim.x, gridDim.y);
        printf("Blocco: %d, %d\n", blockDim.x, blockDim.y);
        printf("\n");

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        WMMAF16TensorCore<<<gridDim, blockDim>>>(mat_a_dev, mat_b_dev, mat_res_dev, sizes[i]);
        hipEventRecord(stop);

        hipMemcpy(mat_res_host_gpu, mat_res_dev, nBytes, hipMemcpyDeviceToHost);

        bool check = true;
#pragma omp parallel for
        for (int k = 0; k < sizes[i] * sizes[i]; k++) {
            if (mat_res_host_gpu[i] != sizes[i])
                check = false;
        }

        printf("Matrix size: %d x %d \n", sizes[i], sizes[i]);
        printf("Check: ");
        if (check) {
            PRINT_GREEN("Verified\n");
        } else {
            PRINT_RED("Error\n");
        }

        //  printMat(mat_res_host_gpu, sizes[i]);
        float elapsed;
        hipEventElapsedTime(&elapsed, start, stop);
        time_stats(elapsed);
        printf("TFLOPS: %.2f\n", static_cast<double>((static_cast<double>(sizes[i]) * sizes[i] * sizes[i] * 2) / (elapsed / 1000.)) / 1e12);

        free(mat_a_host);
        free(mat_b_host);

        hipFree(mat_a_dev);
        hipFree(mat_b_dev);
        hipFree(mat_res_dev);
    }

    return 0;
}