#include <assert.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>

#define WARP_SIZE 32
#define BLOCK_DIM 16

#define PRINT_GREEN(str) printf("\x1b[32m%s\x1b[0m", str);
#define PRINT_RED(str) printf("\x1b[31m%s\x1b[0m", str);

// MMA matrix tile dimensions.
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

using namespace nvcuda;

void time_stats(float micro_seconds) {
    printf("Execution times:\n");
    printf("    * %.0f μs \n", micro_seconds);
    printf("    * %.2f ms \n", micro_seconds / 1000);
    printf("    * %.2f s \n", micro_seconds / 1000 / 1000);
    printf("\n");
}

void printMat(float* mat, int size) {
    // Print the entire matrix
    printf("\n");
    for (int i = 0; i < (size * size); i++) {
        printf("|");
        printf("%05.2f", mat[i]);
        if (((i + 1) % (size) == 0) && (i != 0))
            printf("|\n");
        if ((size * size) == 1)
            printf("|\n");
        if (size == 1 && ((i == 0)))
            printf("|\n");
    }
    printf("\n");
}


__global__ void WMMAF16TensorCore(half* mat_a, half* mat_b, float* mat_c, int size) {
    // Tile using a 2D grid
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // Loop over k
    for (int i = 0; i < size; i += WMMA_K) {
        int aCol = i;
        int aRow = warpM * WMMA_M;
        int bCol = warpN * WMMA_N;
        int bRow = i;

        // Bounds checking
        if (aRow < size && aCol < size && bRow < size && bCol < size) {
            // Load the inputs
            wmma::load_matrix_sync(a_frag, mat_a + aCol + aRow * size, size);
            wmma::load_matrix_sync(b_frag, mat_b + bRow + bCol * size, size);

            // Perform the matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    // Load in the current value of c, scale it by beta, and add this our result
    // scaled by alpha
    int cCol = warpN * WMMA_N;
    int cRow = warpM * WMMA_M;

    if (cRow < size && cCol < size) {
        wmma::load_matrix_sync(c_frag, mat_c + cCol + cRow * size, size, wmma::mem_row_major);

        for (int i = 0; i < c_frag.num_elements; i++) {
            c_frag.x[i] = acc_frag.x[i] + c_frag.x[i];
        }

        // Store the output
        wmma::store_matrix_sync(mat_c + cCol + cRow * size, c_frag, size, wmma::mem_row_major);
    }
}

int main(void) {
    int sizes[1] = {1024};

    half *mat_a_host, *mat_b_host;
    float* mat_res_host_gpu;
    half *mat_a_dev, *mat_b_dev;
    float* mat_res_dev;
    dim3 gridDim, blockDim;

    for (int i = 0; i < 5; i++) {
        long nBytes = sizes[i] * sizes[i] * sizeof(float);

        mat_a_host = (half*)malloc(nBytes);
        mat_b_host = (half*)malloc(nBytes);
        mat_res_host_gpu = (float*)malloc(nBytes);

        hipMalloc((void**)&mat_a_dev, nBytes);
        hipMalloc((void**)&mat_b_dev, nBytes);
        hipMalloc((void**)&mat_res_dev, nBytes);

        for (int j = 0; j < sizes[i] * sizes[i]; j++) {
            mat_a_host[j] = __float2half(1);
            mat_b_host[j] = __float2half(1);
        }

        hipMemcpy(mat_a_dev, mat_a_host, nBytes, hipMemcpyDefault);
        hipMemcpy(mat_b_dev, mat_b_host, nBytes, hipMemcpyDefault);
        hipMemset(mat_res_dev, 0, nBytes);

        blockDim.x = 128;
        blockDim.y = 4;
        gridDim.x = (sizes[i] + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
        gridDim.y = (sizes[i] + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        WMMAF16TensorCore<<<gridDim, blockDim>>>(mat_a_dev, mat_b_dev, mat_res_dev, sizes[i]);
        hipEventRecord(stop);

        hipMemcpy(mat_res_host_gpu, mat_res_dev, nBytes, hipMemcpyDeviceToHost);

        bool check = true;
        for (int k = 0; k < sizes[i] * sizes[i]; k++) {
            if (mat_res_host_gpu[i] != sizes[i])
                check = false;
        }

        printf("Matrix size: %d x %d \n", sizes[i], sizes[i]);
        printf("Block size: %d x %d = %d\n", BLOCK_DIM, BLOCK_DIM, BLOCK_DIM * BLOCK_DIM);
       
        
        printf("Check: ");
        if (check) {
            PRINT_GREEN("Verified\n");
        } else {
            PRINT_RED("Error\n");
        }

         printMat(mat_res_host_gpu, sizes[i]);
        float elapsed;
        hipEventElapsedTime(&elapsed, start, stop);
        time_stats(elapsed);
        printf("TFLOPS: %.2f\n", static_cast<double>((static_cast<double>(sizes[i]) * sizes[i] * sizes[i] * 2) / (elapsed / 1000.)) / 1e12);

        free(mat_a_host);
        free(mat_b_host);

        hipFree(mat_a_dev);
        hipFree(mat_b_dev);
        hipFree(mat_res_dev);
    }

    return 0;
}