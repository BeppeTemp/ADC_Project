#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define SIZE 4

int main(void) {
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed;

    float *a_host, *b_host;
    float *a_dev, *b_dev;
    float res = 0;

    a_host = (float *)malloc(SIZE * sizeof(*a_host));
    b_host = (float *)malloc(SIZE * sizeof(*b_host));

    if (!b_host || !a_host) {
        printf("host memory allocation failed");
        return EXIT_FAILURE;
    }

    // Inizializzo i dati
    srand((unsigned int)time(0));
    for (int i = 0; i < SIZE; i++) {
        a_host[i] = rand() % 5 - 2;
        b_host[i] = rand() % 5 - 2;
    }

    // Alloco a_dev e b_dev
    cudaStat = hipMalloc((void **)&a_dev, SIZE * sizeof(*a_host));
    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed");
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc((void **)&b_dev, SIZE * sizeof(*b_host));
    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed");
        return EXIT_FAILURE;
    }

    // Creo l'handle per cublas
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    // Setto a_host su a_dev
    stat = hipblasSetVector(SIZE, sizeof(float), a_host, 1, a_dev, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("data download failed");
        hipFree(a_dev);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    // Setto b_host su b_dev
    stat = hipblasSetVector(SIZE, sizeof(float), b_host, 1, b_dev, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("data download failed");
        hipFree(b_dev);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    hipEventRecord(start, 0);

    // Calcolo il prodotto
    stat = hipblasSdot(handle, SIZE, a_dev, 1, b_dev, 1, &res);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("data download failed hipblasSdot");
        hipFree(a_dev);
        hipFree(b_dev);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);

    printf("Tempo GPU=%f\n\n", elapsed);
    printf("Risultato del prodotto: %f\n", res);

    hipFree(a_dev);
    hipFree(b_dev);
    hipblasDestroy(handle);
    free(a_host);
    free(b_host);

    return EXIT_SUCCESS;
}